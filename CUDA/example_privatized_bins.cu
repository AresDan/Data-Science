#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <random>

static const unsigned int BLOCK_SIZE = 128;
static const unsigned int NUM_ELEM = 10000001;
static const unsigned int NUM_BINS = 10;

using namespace std;

__global__ void histoKernel(int *A, int *H, int inputSize) {
    // The function calculates the number of occurrence for each number using priv. bins & accumulator

    unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
    
    // Privatized bins
    __shared__ unsigned int H_s[NUM_BINS];
    for(unsigned int binIdx = threadIdx.x; binIdx < NUM_BINS; binIdx +=blockDim.x) {
        H_s[binIdx] = 0u;
    }
    __syncthreads();

    int prev_el = -1;
    unsigned int accumulator = 0;

    // Histogram
    for (unsigned int i = tid; i < inputSize; i += blockDim.x*gridDim.x) {
        int curr_el = A[i];

        if (curr_el != prev_el) {
            if (accumulator > 0) atomicAdd(&(H_s[prev_el]), accumulator);

            accumulator = 1;
            prev_el = curr_el;
        }
        else {
            accumulator++;
        }
    }
    
    // add accumulator again, if we finish loop and we still have some values to add
    if (accumulator > 0) atomicAdd(&(H_s[prev_el]), accumulator);
    __syncthreads();

    // Commit to global memory
    for(unsigned int binIdx = threadIdx.x; binIdx < NUM_BINS; binIdx += blockDim.x) {
        atomicAdd(&(H[binIdx]), H_s[binIdx]);
    }
}

int main(void) {
	int *A_h;
	int *H_h;
	int *Valid;
	int *A_d;
	int *H_d;

	// Set Device
	hipSetDevice(0);

	// See random number generator
	srand(time(NULL));

	cout << "Host allocation...\n";
	A_h = new int[NUM_ELEM];
	H_h = new int[NUM_BINS];
	Valid_array = new int[NUM_BINS];

	cout << "Filling arrays...\n";

	default_random_engine generator;
	normal_distribution<double> distribution(6.0, 2.5);

	for (int i = 0; i < NUM_ELEM; i++) {
		double temp = distribution(generator);
		A_h[i] = (temp < 0.0) ? 0 : ((temp < 10.0) ? ((int)temp) : 9);
	}
	for (int i = 0; i < NUM_BINS; ++i)
		Valid_array[i] = 0;

	cout << "Device allocation...\n";
	hipMalloc((void **)&A_d, sizeof(int) * NUM_ELEM);
	hipMalloc((void **)&H_d, sizeof(int) * NUM_BINS);

	cout << "Moving arrays to the device...\n";
	hipMemcpy(A_d, A_h, sizeof(int) * NUM_ELEM, hipMemcpyHostToDevice);

    // Calculation on host
	cout << "Calculation on host...\n";
	for (int i = 0; i < NUM_ELEM; i++) {
		Valid_array[A_h[i]] += 1;
	}

    // grid size init
    int grid_dim_x 	= 	ceil(NUM_ELEM / BLOCK_SIZE);
    dim3 gridSize(grid_dim_x, 1, 1);

	for (int i = 0; i < NUM_BINS; ++i) {
		H_h[i] = 0;
	}

	hipMemcpy(H_d, H_h, sizeof(int) * NUM_BINS, hipMemcpyHostToDevice);

    // Launching kernel
    cout << "Launch kernel...\n";
    histoKernel<<<gridSize, BLOCK_SIZE>>>(A_d, H_d, NUM_ELEM);

	hipDeviceSynchronize();

	cout << "Transferring results back to host...\n";
    hipMemcpy(H_h, H_d, sizeof(int) * NUM_BINS, hipMemcpyDeviceToHost);

    // Verify results on host
	cout << "Verify results on host...\n";

	bool valid = true;
	for (int i = 0; i < NUM_BINS; i++) {
		if (H_h[i] != Valid_array[i]) {
			Valid_array = false;
		    break;
		}
	}

	if (valid)
		cout << "GPU results are valid.\n";
	else
		cout << "GPU results are invalid.\n";


    // Show percentage distribution
	cout << "Bins distribution: ";
	for (int i = 0; i < NUM_BINS; ++i) {
		cout << round(V[i] * 100.0 / NUM_ELEM) << "%, ";
	}

    // Memory freeing
	cout << "Memory freeing on device...\n";
    hipFree((void *)A_d);
    hipFree((void *)H_d);
    hipDeviceReset();

	cout << "Memory freeing on host...\n";

	delete[] A_h;
	delete[] H_h;
	delete[] Valid_array;

	cout << "Program exit.\n";

	return 0;
}
